#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CudaKmeans.h"
#include <time.h>

void calculateClustersDiameterCuda(Point* points, int numOfPoints, Cluster* clusters, int numOfClusters) {
#pragma omp parallel for
	for (int clusterIndex = 0; clusterIndex < numOfClusters; clusterIndex++) {
		double maxDistance = 0, currDistance;

		// Going over all points and calculating distance with each other point in the cluster
		// to get the maximum distance
		for (int i = 0; i < numOfPoints - 1; i++)
			if (points[i].cluster->id == clusterIndex)
				for (int j = i + 1; j < numOfPoints; j++)
					// Calculating distance for points in the same cluster
					if (points[j].cluster->id == clusterIndex) {
						currDistance = getPointsDistance(&(points[i].position), &(points[j].position));
						if (currDistance > maxDistance)
							maxDistance = currDistance;
					}

		clusters[clusterIndex].diameter = maxDistance;
	}
}

__global__ void increaseTimeKernel(Point *dev_pointArr, double timeInterval, int numOfPoints) {
	// Starting from the thread's id
	// increasing the point index by grid size
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < numOfPoints; i += blockDim.x * gridDim.x)
	{
		dev_pointArr[i].position.x += timeInterval * dev_pointArr[i].velocity.vx;
		dev_pointArr[i].position.y += timeInterval * dev_pointArr[i].velocity.vy;
	}
}

// Increases each point in the given collection by time with the given velocities
// (for each point x = x + (dt * moment) * vxi ,
// y = y + (dt * moment) * vyi)
const char* increaseTimeCudaStart(Point* points, int numOfPoints, double dt, int moment, Point** gpu_points) {
	double timeInterval = dt * moment;
	Point *dev_points = 0;
	int numOfBlocks;
	hipError_t cudaStatus;

	numOfBlocks = (numOfPoints + BLOCK_SIZE - 1) / BLOCK_SIZE;
	
	if (numOfBlocks > MAX_BLOCKS)
		numOfBlocks = MAX_BLOCKS;

	if (*gpu_points == NULL) {
		// Allocate GPU buffers for the points array
		cudaStatus = hipMalloc((void**)&dev_points, numOfPoints * sizeof(Point));
		if (cudaStatus != hipSuccess)
			return "hipMalloc failed!";
		
		*gpu_points = dev_points;
	}
	else
		dev_points = *gpu_points;

	// Copy data array from host memory to GPU buffers
	cudaStatus = hipMemcpy(dev_points, points, numOfPoints * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		hipFree(dev_points);
		return "hipMemcpy failed!";
	}

	// Launch a kernel increasing time for one part of the points on the GPU with
	increaseTimeKernel << <numOfBlocks, BLOCK_SIZE >> >(dev_points, timeInterval, numOfPoints);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		hipFree(dev_points);
		return "countKernel launch failed";
	}

	return NULL;
}

const char* increaseTimeCudaEnd(Point* dev_points, Point* pointsArr, int numOfPoints) {
	hipError_t cudaStatus;

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		//hipFree(&dev_data, &dev_threadsCounterArray, &dev_histogram);
		return "hipDeviceSynchronize returned error code after launching countKernel!";
	}

	// Copy histogram result vector from GPU buffer to host memory
	cudaStatus = hipMemcpy(pointsArr, dev_points, numOfPoints * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		//hipFree(&dev_data, &dev_threadsCounterArray, &dev_histogram);
		return "hipMemcpy failed!";
	}

	// Free all GPU memory
	//hipFree(&dev_data, &dev_threadsCounterArray, &dev_histogram);
	return NULL;
}

const char* stopCuda() {
	hipError_t cudaStatus;

	// was at main at first, need to be checked 
	// hipDeviceReset must be called before exiting
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		//hipFree(&dev_data, &dev_threadsCounterArray, &dev_histogram);
		return "hipDeviceReset failed";
	}

	return NULL;
}

const char* initCuda() {
	hipError_t cudaStatus;

	// Choose which GPU to run on (our system has only one)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		//hipFree(&dev_data, &dev_threadsCounterArray, &dev_histogram);
		return "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
	}

	return NULL;
}

const char* freeCuda(Point* gpu_points) {
	if (hipFree(gpu_points) != hipSuccess)
		return "hipFree error";
	return NULL;
}

void calculateClustersCentersCuda(Point* points, int numOfPoints, Cluster* clusters, int numOfClusters) {
#pragma omp parallel for
	for (int clusterIndex = 0; clusterIndex < numOfClusters; clusterIndex++) {
		double sumY = 0, sumX = 0;

		// When a cluster has no points, we keep it's center intact
		if (clusters[clusterIndex].numOfPoints != 0) {

			for (int pointIndex = 0; pointIndex < numOfPoints; pointIndex++) {
				// Calculating sum of all point in the cluster
				if (points[pointIndex].cluster->id == clusterIndex) {
					sumX += points[pointIndex].position.x;
					sumY += points[pointIndex].position.y;
				}
			}

			// Each cluster's center is the average of points positions
			clusters[clusterIndex].center.x = sumX / clusters[clusterIndex].numOfPoints;
			clusters[clusterIndex].center.y = sumY / clusters[clusterIndex].numOfPoints;
		}
	}
}

// Assigning all points to clusters
// Returns: TRUE if the point's cluster has changed, FALSE if not
Boolean assignClustersToPointsCuda(Point* points, int numOfPoints, Cluster* clusters, int numOfClusters) {
	int pointIndex;
	// Indicating if at least one point has changed cluster
	Boolean pointChanged = FALSE;

#pragma omp parallel for
	for (pointIndex = 0; pointIndex < numOfPoints; pointIndex++)
		// Assigning the curr point and checking if changed cluster
		if (assignClusterToPoint(&(points[pointIndex]), clusters, numOfClusters))
			pointChanged = TRUE;

	return pointChanged;
}
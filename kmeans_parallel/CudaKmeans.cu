#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CudaKmeans.h"
#include <time.h>

// Calling device reset
const char* stopCuda() {
	hipError_t cudaStatus;

	// was at main at first, need to be checked 
	// hipDeviceReset must be called before exiting
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
		return "hipDeviceReset failed";

	return NULL;
}

// Setting the cuda device (0)
const char* initCuda() {
	hipError_t cudaStatus;

	// Choose which GPU to run on (our system has only one)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
		return "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";

	return NULL;
}

__global__ void increaseTimeKernel(Point *dev_pointArr, double timeInterval, int numOfPoints) {
	// Starting from the thread's id
	// increasing the point index by grid size
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < numOfPoints; i += blockDim.x * gridDim.x)
	{
		dev_pointArr[i].position.x += timeInterval * dev_pointArr[i].velocity.vx;
		dev_pointArr[i].position.y += timeInterval * dev_pointArr[i].velocity.vy;
	}
}

// Increases each point in the given collection by time with the given velocities
// (for each point x = x + (dt * moment) * vxi ,
// y = y + (dt * moment) * vyi)
const char* increaseTimeCudaStart(Point* points, int numOfPoints, double dt, int moment, Point** gpu_points) {
	double timeInterval = dt * moment;
	Point *dev_points = 0;
	int numOfBlocks;
	hipError_t cudaStatus;

	numOfBlocks = (numOfPoints + BLOCK_SIZE - 1) / BLOCK_SIZE;
	
	if (numOfBlocks > MAX_BLOCKS)
		numOfBlocks = MAX_BLOCKS;

	if (*gpu_points == NULL) {
		// Allocate GPU buffers for the points array
		cudaStatus = hipMalloc((void**)&dev_points, numOfPoints * sizeof(Point));
		if (cudaStatus != hipSuccess)
			return "hipMalloc failed!";
		
		*gpu_points = dev_points;
	}
	else
		dev_points = *gpu_points;

	// Copy data array from host memory to GPU buffers
	cudaStatus = hipMemcpy(dev_points, points, numOfPoints * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		hipFree(dev_points);
		return "hipMemcpy failed!";
	}

	// Launch a kernel increasing time for one part of the points on the GPU with
	increaseTimeKernel << <numOfBlocks, BLOCK_SIZE >> >(dev_points, timeInterval, numOfPoints);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		hipFree(dev_points);
		return "countKernel launch failed";
	}

	return NULL;
}

const char* increaseTimeCudaEnd(Point* dev_points, Point* pointsArr, int numOfPoints) {
	hipError_t cudaStatus;

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		hipFree(dev_points);
		return "hipDeviceSynchronize returned error code after launching countKernel!";
	}

	// Copy histogram result vector from GPU buffer to host memory
	cudaStatus = hipMemcpy(pointsArr, dev_points, numOfPoints * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		hipFree(dev_points);
		return "hipMemcpy failed!";
	}

	// Free all GPU memory
	return NULL;
}

const char* freeCuda(Point* gpu_points) {
	if (hipFree(gpu_points) != hipSuccess)
		return "hipFree error";
	return NULL;
}